#include "hip/hip_runtime.h"
#include "mex.h"
#include "stdio.h"
#include "string.h"
#include <windows.h>
#include <assert.h>
#include <stdlib.h>
#include "instances.h"
#include "MRFEnergy.h"
#include "hip/hip_runtime.h"
#include ""
#include "gpu/mxGPUArray.h"

#include "hip/hip_runtime.h"


void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[]){
    bool IsUseGPU;
    if(nrhs==5)
        IsUseGPU=bool(*((double*)mxGetPr(prhs[4])));
    else if(nrhs==4)
        IsUseGPU=true;
    else
        mexErrMsgTxt("\nErrors in input.\n");
	MRFEnergy<TypeGeneral>* mrf;
	MRFEnergy<TypeGeneral>::NodeId* nodes;
	MRFEnergy<TypeGeneral>::Options options;
	TypeGeneral::REAL energy, lowerBound;

    double* EdgeTerminals=((double*)mxGetPr(prhs[0]));
    int EdgeNum,LabelNum,nodeNum;
    EdgeNum=mxGetN(prhs[0]);
    TypeGeneral::REAL* f1=((double*)mxGetPr(prhs[1]));
    LabelNum=mxGetM(prhs[1]);
    nodeNum=mxGetN(prhs[1]);
    TypeGeneral::REAL* f2=((double*)mxGetPr(prhs[2]));
    TypeGeneral::REAL* op=((double*)mxGetPr(prhs[3]));
    
    int i;
    double tmp=0;
    for(i=0;i<EdgeNum*LabelNum*LabelNum;i++){
        tmp=min(tmp,f2[i]);
    }
    if(tmp<0)
        mexErrMsgTxt("\nErrors: having negative pairwise elements.\n");
    mrf=new MRFEnergy<TypeGeneral>(TypeGeneral::GlobalSize());
	nodes=new MRFEnergy<TypeGeneral>::NodeId[nodeNum];
    mexPrintf("add nodes\n");
	
    for(i=0;i<nodeNum;i++){
        nodes[i]=mrf->AddNode(TypeGeneral::LocalSize(LabelNum), TypeGeneral::NodeData(&f1[LabelNum*i]));
    }
    mexPrintf("add edges\n");
    for(i=0;i<EdgeNum;i++){
        mrf->AddEdge(nodes[int(EdgeTerminals[i*2])],nodes[int(EdgeTerminals[i*2+1])],TypeGeneral::EdgeData(TypeGeneral::GENERAL,&f2[LabelNum*LabelNum*i]));
    }
    mexPrintf("set ordering\n");
	// Function below is optional - it may help if, for example, nodes are added in a random order
	mrf->SetAutomaticOrdering();
	/////////////////////// TRW-S algorithm //////////////////////
	options.m_iterMax=int(op[1]); // maximum number of iterations
    options.m_eps=op[0];
    mexPrintf("energy minimization\n");
	mrf->Minimize_TRW_S(IsUseGPU,options,lowerBound,energy);
    mexPrintf("trws done\n");
	// read solution
    double* x;
    plhs[0]=mxCreateDoubleMatrix(nodeNum,1,mxREAL);
	x=mxGetPr(plhs[0]);
    for(i=0;i<nodeNum;i++){
        x[i]=mrf->GetSolution(nodes[i]);
    }
	delete nodes;
	delete mrf;
}
